#include "hip/hip_runtime.h"
#include <iostream>
#include <stdexcept>
#include <math.h> 
#include <cmath>
#include <mpi.h>
#include <string>
#include <sstream>
#include <fstream>
#include <cstdlib>
#include <ctime>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <sstream>
#include <string>

using namespace std;

#define SAFE_CUDA(err)\
    if (err != hipSuccess){ \
        throw std::runtime_error(hipGetErrorString(err)); \
    }

#define CUDA_CHECK_ERROR\
    if (hipPeekAtLastError() != hipSuccess){ \
        throw std::runtime_error(hipGetErrorString(hipGetLastError())); \
    }

double F(const double x, const double y) {
    double t = 1.0 + 1.0*x*y;
    if (t == 0)
    	throw std::runtime_error("Error in computing 'F' function");
    return (x*x + y*y)/(t*t);
}

double phi(const double x, const double y) {
    double t = 1.0 + 1.0 * x*y;
    if (t <= 0)
        throw std::runtime_error("Error in computing 'phi' function");
    return log(t);
}

__device__ double gpu_F(const double x, const double y) {
    return (x*x + y*y) / ((1.0 + 1.0*x*y)*(1.0 + 1.0*x*y));
}

double f_grid(const double t) {
	double q = 1.5;
	if (t < 0 || t > 1)
		throw std::runtime_error("Error in computing 'f_grid' function");
	return (pow(1.0 + t, q) - 1.0) / (pow(2.0, q) - 1.0);
}

void compute_grid_processes_number(const int& size, int& x_proc_num, int& y_proc_num) {
    if (size >= 512) {
        x_proc_num = 16;
        y_proc_num = 32;
    } else if (size >= 256) {
        x_proc_num = 16;
        y_proc_num = 16;
    } else if (size >= 128) {
        x_proc_num = 8;
        y_proc_num = 16;
    } else if (size >= 64) {
        x_proc_num = 8;
        y_proc_num = 8;
    } else if (size >= 32) {
        x_proc_num = 4;
        y_proc_num = 8;
    } else if (size >= 16) {
        x_proc_num = 4;
        y_proc_num = 4;
    } else if (size >= 8){
        x_proc_num = 2;
        y_proc_num = 4;
    } else if (size >= 4) {
        x_proc_num = 2;
        y_proc_num = 2;
    } else if (size >= 2) {
        x_proc_num = 1;
        y_proc_num = 2;
    } else if (size >= 1) {
        x_proc_num = 1;
        y_proc_num = 1;
    } else {
        throw std::runtime_error("Incorrect processes number");
    }
}

struct GridParameters {
	int rank, N1, N2, p1, p2, x_index_from, x_index_to, y_index_from, y_index_to;
	double *x_grid, *y_grid;
	double eps;
    bool top, bottom, left, right;
    double *hxhy, *gp_x_grid, *gp_y_grid, *gp_is_not_border, *gp_x_h_step, *gp_y_h_step;

    double *send_message_top, *send_message_bottom, *send_message_left, *send_message_right;
    double *recv_message_top, *recv_message_bottom, *recv_message_left, *recv_message_right;
    MPI_Request* send_requests;
    MPI_Request* recv_requests;
    MPI_Comm comm;

	GridParameters (int rank, MPI_Comm comm, double* x_grid, double* y_grid, int N1, int N2, int p1, int p2, double eps):
		rank (rank), comm (comm), x_grid (x_grid), y_grid (y_grid), 
		send_message_top (NULL), send_message_bottom (NULL), send_message_left (NULL), send_message_right (NULL),
		recv_message_top (NULL), recv_message_bottom (NULL), recv_message_left (NULL), recv_message_right (NULL),
		send_requests (NULL), recv_requests (NULL),
		N1 (N1), N2 (N2),p1 (p1), p2 (p2), eps (eps), 
		x_index_from (0), x_index_to (0), y_index_from (0), y_index_to (0),
		top (false), bottom (false), left (false), right (false) {
			int step1, step2;
			step1 = int(floor(1.0 * N1 / p1));
			step2 = int(floor(1.0 * N2 / p2));
			x_index_from = int(floor(1.0 * step1 * floor(1.0 * rank / p2)));
			y_index_from = int(floor((double(rank % p2)) * step2));

			if ((rank + 1) % p2 == 0)
				y_index_to = N2;
			else
				y_index_to = y_index_from + step2; 

			if (rank >= (p1-1)*p2)
				x_index_to = N1;
			else
				x_index_to = x_index_from + step1;

			if (x_index_from == 0)
				top = true;
			if (y_index_from == 0)
				left = true;
			if (y_index_to == N1)
				right = true;
			if (x_index_to == N1)
				bottom = true;

			hxhy = new double [get_num_x_points() * get_num_y_points()];
			for (int i=0; i<get_num_x_points(); i++){
	        	for (int j=0; j<get_num_y_points(); j++){
	        		int grid_i, grid_j;
    				get_real_grid_index(i, j, grid_i, grid_j);
    				if (not is_border_point(grid_i, grid_j)) 
	        			hxhy[i*get_num_y_points()+j] = ((get_x_h_step(grid_i) + get_x_h_step(grid_i-1)) / 2.0) * ((get_y_h_step(grid_j) + get_y_h_step(grid_j-1)) / 2.0);
	        		else
	        			hxhy[i*get_num_y_points()+j] = 0.0;
	        	}
	        }

	        gp_x_grid = new double [get_num_x_points() * get_num_y_points()];
			gp_y_grid = new double [get_num_x_points() * get_num_y_points()];
			gp_is_not_border = new double [get_num_x_points() * get_num_y_points()];
			gp_x_h_step = new double [get_num_x_points() * get_num_y_points()];
			gp_y_h_step = new double [get_num_x_points() * get_num_y_points()];

			for (int i=0; i<get_num_x_points(); i++) {
		    	for (int j=0; j<get_num_y_points(); j++) {
		    		int grid_i, grid_j;
		    		get_real_grid_index(i, j, grid_i, grid_j);
		    		gp_x_grid[i*get_num_y_points()+j] = get_x_grid_value(grid_i);
		    		gp_y_grid[i*get_num_y_points()+j] = get_y_grid_value(grid_j);
		    		if (is_border_point(grid_i, grid_j))
						gp_is_not_border[i*get_num_y_points()+j] = 0.0;
					else
						gp_is_not_border[i*get_num_y_points()+j] = 1.0;
					if ((i < get_num_x_points() - 1) && (j < get_num_y_points() - 1)) {
						gp_x_h_step[i*get_num_y_points()+j] = get_x_h_step(grid_i);
						gp_y_h_step[i*get_num_y_points()+j] = get_y_h_step(grid_j);
					}
		    	}
			}
		}

	int get_num_x_points() {
		if (bottom) 
			return x_index_to - x_index_from + 1;
		else
			return x_index_to - x_index_from;
	}

	int get_num_y_points() {
		if (right) 
			return y_index_to - y_index_from + 1;
		else
			return y_index_to - y_index_from;
	}

	void get_real_grid_index(int i, int j, int& grid_i, int& grid_j) {
		grid_i = x_index_from+i;
		grid_j = y_index_from+j;
	}

	double get_x_grid_value(int grid_i) {
		return x_grid[grid_i];
	}

	double get_y_grid_value(int grid_j) {
		return y_grid[grid_j];
	}

	double get_x_h_step(int grid_i) {
		return x_grid[grid_i+1] - x_grid[grid_i];
	}

	double get_y_h_step(int grid_j) {
		return y_grid[grid_j+1] - y_grid[grid_j];
	}

	int get_top_rank() {
		return rank - p2;
	}

	int get_bottom_rank() {
		return rank + p2;
	}

	int get_left_rank() {
		return rank - 1;
	}

	int get_right_rank() {
		return rank + 1;
	}

	bool is_border_point(int grid_i, int grid_j) {
		if ((grid_i == 0) || (grid_j == 0) || (grid_i == N1) || (grid_j == N2))
			return true;
		else
			return false;
	}
};

__global__ void gpu_scalar_product(double *f1, double *f2, double* hxhy, double *results, int n) {
	extern __shared__ double sdata[];
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	// load input into __shared__ memory
	double x = 0.0;
	if(i < n)
		x = hxhy[i] * f1[i] * f2[i];//input[i];

	sdata[tx] = x;
	__syncthreads(); 
	// block-wide reduction in __shared__ mem
	for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
		if(tx < offset) {
			// add a partial sum upstream to our own
			sdata[tx] += sdata[tx + offset];
		}
		__syncthreads();
	}
	// finally, thread 0 writes the result
	if(threadIdx.x == 0) {
		// note that the result is per-block
		// not per-thread
		results[blockIdx.x] = sdata[0];
	}
}


__global__ void gpu_reduce_sum(double *input, double *results, int n) {
	extern __shared__ double sdata[];
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	// load input into __shared__ memory
	double x = 0.0;
	if(i < n)
		x = input[i];

	sdata[tx] = x;
	__syncthreads(); 
	// block-wide reduction in __shared__ mem
	for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
		if(tx < offset) {
			// add a partial sum upstream to our own
			sdata[tx] += sdata[tx + offset];
		}
		__syncthreads();
	}
	// finally, thread 0 writes the result
	if(threadIdx.x == 0) {
		// note that the result is per-block
		// not per-thread
		results[blockIdx.x] = sdata[0];
	}
}


double scalar_product(GridParameters gp, const double* f1, const double* f2) {
 	int size = gp.get_num_x_points() * gp.get_num_y_points() * sizeof(double);
    int numElements = gp.get_num_x_points() * gp.get_num_y_points();

	double *d_f1, *d_f2, *d_hxhy, *d_product;
    SAFE_CUDA(hipMalloc(&d_f1, size));
    SAFE_CUDA(hipMalloc(&d_f2, size));
    SAFE_CUDA(hipMalloc(&d_hxhy, size));

    SAFE_CUDA(hipMemcpy(d_f1, f1, size, hipMemcpyHostToDevice));
    SAFE_CUDA(hipMemcpy(d_f2, f2, size, hipMemcpyHostToDevice));
    SAFE_CUDA(hipMemcpy(d_hxhy, gp.hxhy, size, hipMemcpyHostToDevice));

	int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	SAFE_CUDA(hipMalloc(&d_product, (blocksPerGrid+1)*sizeof(double)));
    //printf("rank=%d CUDA kernel launch with %d blocks of %d threads numElements=%d\n", gp.rank, blocksPerGrid, threadsPerBlock, numElements);
    // reduce per-block partial sums 
    gpu_scalar_product<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(double)>>>(d_f1, d_f2, d_hxhy, d_product, numElements); CUDA_CHECK_ERROR;

    // reduce partial sums to a total sum
    gpu_reduce_sum<<<1, threadsPerBlock, threadsPerBlock * sizeof(double)>>>(d_product, d_product+blocksPerGrid, blocksPerGrid); CUDA_CHECK_ERROR;

    double gpu_product = 0.0;
    SAFE_CUDA(hipMemcpy(&gpu_product, d_product+blocksPerGrid, sizeof(double), hipMemcpyDeviceToHost));
    //printf("rank=%d size=%d numElements=%d blocksPerGrid=%d CPU product=%f GPU product=%f\n", 
    //	gp.rank, size, numElements, blocksPerGrid, product, gpu_product);

    SAFE_CUDA(hipFree(d_f1));
    SAFE_CUDA(hipFree(d_f2));
    SAFE_CUDA(hipFree(d_hxhy));
    SAFE_CUDA(hipFree(d_product));

    double global_product = 0.0;
    int status = MPI_Allreduce(&gpu_product, &global_product, 1, MPI_DOUBLE, MPI_SUM, gp.comm);
    if (status != MPI_SUCCESS) throw std::runtime_error("Error in compute scalar_product!");
    //printf("rank %d: product=%f global_product=%f\n", gp.rank, product, global_product);
    return global_product;
}

void compute_delta(GridParameters gp, const double *func, double *delta_func, double f_top, double f_bottom, double f_left, double f_right, int i, int j, int grid_i, int grid_j) {
	double h_i_1 = gp.get_x_h_step(grid_i-1);
	double h_i = gp.get_x_h_step(grid_i);
	double h_j_1 = gp.get_y_h_step(grid_j-1);
	double h_j = gp.get_y_h_step(grid_j);
	double average_hx = (h_i + h_i_1) / 2.0;
	double average_hy = (h_j + h_j_1) / 2.0;
	double f_curr = func[i*gp.get_num_y_points()+j];
	delta_func[i*gp.get_num_y_points()+j] = 
		(1.0 / average_hx) * ((f_curr - f_top) / h_i_1 - (f_bottom - f_curr) / h_i) + 
		(1.0 / average_hy) * ((f_curr - f_left) / h_j_1 - (f_right - f_curr) / h_j);
	//printf("i=%d j=%d grid_i=%d grid_j=%d average_hx=%f average_hy=%f h_i_1=%f h_i=%f h_j_1=%f h_j=%f f_curr=%f f_top=%f f_bottom=%f f_left=%f f_right=%f delta_func[i][j] = %f\n", i, j, grid_i, grid_j, average_hx, average_hy, h_i_1, h_i, h_j_1, h_j, f_curr, f_top, f_bottom, f_left, f_right, delta_func[i*gp.get_num_y_points()+j]);
}

__global__ void gpu_compute_approx_delta(double *delta_func, double *func, double *gp_x_h_step, double* gp_y_h_step, 
	double* gp_is_not_border, int y_shape, int numElements) {
  int no_thread = threadIdx.x + blockDim.x * blockIdx.x;

  if ((no_thread < numElements) && (gp_is_not_border[no_thread] == 1.0)) {
  	double h_i_1 = gp_x_h_step[no_thread-1];
	double h_i = gp_x_h_step[no_thread];
	double h_j_1 = gp_y_h_step[no_thread-1];
	double h_j = gp_y_h_step[no_thread];
  	double average_hx = (h_i + h_i_1) / 2.0;
	double average_hy = (h_j + h_j_1) / 2.0;
	double f_curr = func[no_thread];
	double f_top = func[no_thread-y_shape];
	double f_bottom = func[no_thread+y_shape];
	double f_left = func[no_thread-1];
	double f_right = func[no_thread+1];
  	delta_func[no_thread] = (1.0 / average_hx) * ((f_curr - f_top) / h_i_1 - (f_bottom - f_curr) / h_i) + 
  			(1.0 / average_hy) * ((f_curr - f_left) / h_j_1 - (f_right - f_curr) / h_j);
  }
}

enum MPI_tags { SendToTop, SendToBottom, SendToLeft, SendToRight};

void compute_approx_delta(GridParameters gp, double* delta_func, const double* func) {
	int i, j;

	int size = gp.get_num_x_points() * gp.get_num_y_points() * sizeof(double);
    int numElements = gp.get_num_x_points() * gp.get_num_y_points();

	double *d_delta_func, *d_func, *d_gp_x_h_step, *d_gp_y_h_step, *d_gp_is_not_border;
	SAFE_CUDA(hipMalloc(&d_delta_func, size));
	SAFE_CUDA(hipMalloc(&d_func, size));
	SAFE_CUDA(hipMalloc(&d_gp_x_h_step, size));
	SAFE_CUDA(hipMalloc(&d_gp_y_h_step, size));
	SAFE_CUDA(hipMalloc(&d_gp_is_not_border, size));

	SAFE_CUDA(hipMemcpy(d_delta_func, delta_func, size, hipMemcpyHostToDevice));
	SAFE_CUDA(hipMemcpy(d_func, func, size, hipMemcpyHostToDevice));
	SAFE_CUDA(hipMemcpy(d_gp_x_h_step, gp.gp_x_h_step, size, hipMemcpyHostToDevice));
	SAFE_CUDA(hipMemcpy(d_gp_y_h_step, gp.gp_y_h_step, size, hipMemcpyHostToDevice));
	SAFE_CUDA(hipMemcpy(d_gp_is_not_border, gp.gp_is_not_border, size, hipMemcpyHostToDevice));

	int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    gpu_compute_approx_delta<<<blocksPerGrid, threadsPerBlock>>>(d_delta_func, d_func, d_gp_x_h_step, d_gp_y_h_step, d_gp_is_not_border, gp.get_num_y_points(), numElements); CUDA_CHECK_ERROR;

    SAFE_CUDA(hipMemcpy(delta_func, d_delta_func, size, hipMemcpyDeviceToHost));

    SAFE_CUDA(hipFree(d_delta_func));
    SAFE_CUDA(hipFree(d_func));
    SAFE_CUDA(hipFree(d_gp_x_h_step));
    SAFE_CUDA(hipFree(d_gp_y_h_step));
    SAFE_CUDA(hipFree(d_gp_is_not_border));

	if (gp.send_message_top == NULL)
		gp.send_message_top = new double [gp.get_num_y_points()];
	if (gp.send_message_bottom == NULL)
		gp.send_message_bottom = new double [gp.get_num_y_points()];
	if (gp.send_message_left == NULL)
		gp.send_message_left = new double [gp.get_num_x_points()];
	if (gp.send_message_right == NULL)
		gp.send_message_right = new double [gp.get_num_x_points()];

	if (gp.recv_message_top == NULL)
		gp.recv_message_top = new double [gp.get_num_y_points()];
	if (gp.recv_message_bottom == NULL)
		gp.recv_message_bottom = new double [gp.get_num_y_points()];
	if (gp.recv_message_left == NULL)
		gp.recv_message_left = new double [gp.get_num_x_points()];
	if (gp.recv_message_right == NULL)
		gp.recv_message_right = new double [gp.get_num_x_points()];

	if (gp.send_requests == NULL)
		gp.send_requests = new MPI_Request [4];
	if (gp.recv_requests == NULL)
		gp.recv_requests = new MPI_Request [4];

	for (int j=0; j<gp.get_num_y_points(); j++)
		gp.send_message_top[j] = func[0*gp.get_num_y_points()+j];
	for (int j=0; j<gp.get_num_y_points(); j++)
		gp.send_message_bottom[j] = func[(gp.get_num_x_points()-1)*gp.get_num_y_points()+j];
	for (int i=0; i<gp.get_num_x_points(); i++)
		gp.send_message_left[i] = func[i*gp.get_num_y_points()+0];
	for (int i=0; i<gp.get_num_x_points(); i++)
		gp.send_message_right[i] = func[i*gp.get_num_y_points()+gp.get_num_y_points()-1];

	int status;
	int send_count=0;
	if (not gp.top) {
		status = MPI_Isend(gp.send_message_top, gp.get_num_y_points(), MPI_DOUBLE, 
			gp.get_top_rank(), SendToTop, gp.comm, &(gp.send_requests[send_count]));
		if (status != MPI_SUCCESS) throw std::runtime_error("Error in send message!");
		send_count++;
	}
	if (not gp.bottom) {
		status = MPI_Isend(gp.send_message_bottom, gp.get_num_y_points(), MPI_DOUBLE, 
			gp.get_bottom_rank(), SendToBottom, gp.comm, &(gp.send_requests[send_count]));
		if (status != MPI_SUCCESS) throw std::runtime_error("Error in send message!");
		send_count++;
	}
	if (not gp.left) {
		status = MPI_Isend(gp.send_message_left, gp.get_num_x_points(), MPI_DOUBLE, 
			gp.get_left_rank(), SendToLeft, gp.comm, &(gp.send_requests[send_count]));
		if (status != MPI_SUCCESS) throw std::runtime_error("Error in send message!");
		send_count++;
	}
	if (not gp.right) {
		status = MPI_Isend(gp.send_message_right, gp.get_num_x_points(), MPI_DOUBLE, 
			gp.get_right_rank(), SendToRight, gp.comm, &(gp.send_requests[send_count]));
		if (status != MPI_SUCCESS) throw std::runtime_error("Error in send message!");
		send_count++;
	}

	int recv_count=0;
	if (not gp.top) {
		status = MPI_Irecv(gp.recv_message_top, gp.get_num_y_points(), MPI_DOUBLE, 
			gp.get_top_rank(), SendToBottom, gp.comm, &(gp.recv_requests[recv_count]));
		if (status != MPI_SUCCESS) throw std::runtime_error("Error in receive message!");
		recv_count++;
	}
	if (not gp.bottom) {
		status = MPI_Irecv(gp.recv_message_bottom, gp.get_num_y_points(), MPI_DOUBLE, 
			gp.get_bottom_rank(), SendToTop, gp.comm, &(gp.recv_requests[recv_count]));
		if (status != MPI_SUCCESS) throw std::runtime_error("Error in receive message!");
		recv_count++;
	}
	if (not gp.left) {
		status = MPI_Irecv(gp.recv_message_left, gp.get_num_x_points(), MPI_DOUBLE, 
			gp.get_left_rank(), SendToRight, gp.comm, &(gp.recv_requests[recv_count]));
		if (status != MPI_SUCCESS) throw std::runtime_error("Error in receive message!");
		recv_count++;
	}
	if (not gp.right) {
		status = MPI_Irecv(gp.recv_message_right, gp.get_num_x_points(), MPI_DOUBLE, 
			gp.get_right_rank(), SendToLeft, gp.comm, &(gp.recv_requests[recv_count]));
		if (status != MPI_SUCCESS) throw std::runtime_error("Error in receive message!");
		recv_count++;
	}

	status = MPI_Waitall(recv_count, gp.recv_requests, MPI_STATUS_IGNORE);
    if (status != MPI_SUCCESS) throw std::runtime_error("Error in waiting receive message!");

    status = MPI_Waitall(send_count, gp.send_requests, MPI_STATUS_IGNORE);
    if (status != MPI_SUCCESS) throw std::runtime_error("Error in waiting send message!");

    if (not gp.top) {
    	int i = 0;
    	for (int j=1; j<gp.get_num_y_points()-1; j++) {
    		int grid_i, grid_j;
    		gp.get_real_grid_index(i, j, grid_i, grid_j);
    		compute_delta(gp, func, delta_func, gp.recv_message_top[j], func[(i+1)*gp.get_num_y_points()+j], func[i*gp.get_num_y_points()+j-1], func[i*gp.get_num_y_points()+j+1], i, j, grid_i, grid_j);
    	}
    }

	if (not gp.bottom) {
    	int i = gp.get_num_x_points()-1;
    	for (int j=1; j<gp.get_num_y_points()-1; j++) {
    		int grid_i, grid_j;
    		gp.get_real_grid_index(i, j, grid_i, grid_j);
    		compute_delta(gp, func, delta_func, func[(i-1)*gp.get_num_y_points()+j], gp.recv_message_bottom[j], func[i*gp.get_num_y_points()+j-1], func[i*gp.get_num_y_points()+j+1], i, j, grid_i, grid_j);
    	}
    }

    if (not gp.left) {
    	int j = 0;
    	for (int i=1; i<gp.get_num_x_points()-1; i++) {
    		int grid_i, grid_j;
    		gp.get_real_grid_index(i, j, grid_i, grid_j);
    		compute_delta(gp, func, delta_func, func[(i-1)*gp.get_num_y_points()+j], func[(i+1)*gp.get_num_y_points()+j], gp.recv_message_left[i], func[i*gp.get_num_y_points()+j+1], i, j, grid_i, grid_j);
    	}
    }

    if (not gp.right) {
    	int j = gp.get_num_y_points()-1;
    	for (int i=1; i<gp.get_num_x_points()-1; i++) {
    		int grid_i, grid_j;
    		gp.get_real_grid_index(i, j, grid_i, grid_j);
    		compute_delta(gp, func, delta_func, func[(i-1)*gp.get_num_y_points()+j], func[(i+1)*gp.get_num_y_points()+j], func[i*gp.get_num_y_points()+j-1], gp.recv_message_right[i], i, j, grid_i, grid_j);
    	}
    }

    // compute corners
	i = 0; j = 0;
	if (not gp.top && not gp.left) {
		int grid_i, grid_j;
    	gp.get_real_grid_index(i, j, grid_i, grid_j);
    	compute_delta(gp, func, delta_func, gp.recv_message_top[j], func[(i+1)*gp.get_num_y_points()+j], gp.recv_message_left[i], func[i*gp.get_num_y_points()+j+1], i, j, grid_i, grid_j);
	}

	i = 0; j = gp.get_num_y_points()-1;
	if (not gp.top && not gp.right) {
		int grid_i, grid_j;
    	gp.get_real_grid_index(i, j, grid_i, grid_j);
    	compute_delta(gp, func, delta_func, gp.recv_message_top[j], func[(i+1)*gp.get_num_y_points()+j], func[i*gp.get_num_y_points()+j-1], gp.recv_message_right[i], i, j, grid_i, grid_j);
	}

	i = gp.get_num_x_points()-1; j = 0;
	if (not gp.bottom && not gp.left) {
		int grid_i, grid_j;
    	gp.get_real_grid_index(i, j, grid_i, grid_j);
    	compute_delta(gp, func, delta_func, func[(i-1)*gp.get_num_y_points()+j], gp.recv_message_bottom[j], gp.recv_message_left[i], func[i*gp.get_num_y_points()+j+1], i, j, grid_i, grid_j);
	}

	i = gp.get_num_x_points()-1; j = gp.get_num_y_points()-1;
	if (not gp.bottom && not gp.right) {
		int grid_i, grid_j;
    	gp.get_real_grid_index(i, j, grid_i, grid_j);
    	compute_delta(gp, func, delta_func, func[(i-1)*gp.get_num_y_points()+j], gp.recv_message_bottom[j], func[i*gp.get_num_y_points()+j-1], gp.recv_message_right[i], i, j, grid_i, grid_j);
	}
}

__global__ void gpu_compute_r(const double *delta_p, double *gp_x_grid, double *gp_y_grid, double* gp_is_not_border, double* r, int numElements) {
  int no_thread = threadIdx.x + blockDim.x * blockIdx.x;

  if (no_thread < numElements) {
  	r[no_thread] = (delta_p[no_thread] - gpu_F(gp_x_grid[no_thread], gp_y_grid[no_thread])) * gp_is_not_border[no_thread];
  }
}

void compute_r(GridParameters gp, double *r, const double *delta_p) {
	int size = gp.get_num_x_points() * gp.get_num_y_points() * sizeof(double);
    int numElements = gp.get_num_x_points() * gp.get_num_y_points();

	double *d_r, *d_delta_p, *d_gp_x_grid, *d_gp_y_grid, *d_gp_is_not_border;
	SAFE_CUDA(hipMalloc(&d_r, size));
	SAFE_CUDA(hipMalloc(&d_delta_p, size));
	SAFE_CUDA(hipMalloc(&d_gp_x_grid, size));
	SAFE_CUDA(hipMalloc(&d_gp_y_grid, size));
	SAFE_CUDA(hipMalloc(&d_gp_is_not_border, size));

	SAFE_CUDA(hipMemcpy(d_r, r, size, hipMemcpyHostToDevice));
	SAFE_CUDA(hipMemcpy(d_delta_p, delta_p, size, hipMemcpyHostToDevice));
	SAFE_CUDA(hipMemcpy(d_gp_x_grid, gp.gp_x_grid, size, hipMemcpyHostToDevice));
	SAFE_CUDA(hipMemcpy(d_gp_y_grid, gp.gp_y_grid, size, hipMemcpyHostToDevice));
	SAFE_CUDA(hipMemcpy(d_gp_is_not_border, gp.gp_is_not_border, size, hipMemcpyHostToDevice));

	int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    gpu_compute_r<<<blocksPerGrid, threadsPerBlock>>>(d_delta_p, d_gp_x_grid, d_gp_y_grid, d_gp_is_not_border, d_r, numElements); CUDA_CHECK_ERROR;

    SAFE_CUDA(hipMemcpy(r, d_r, size, hipMemcpyDeviceToHost));

    SAFE_CUDA(hipFree(d_r));
    SAFE_CUDA(hipFree(d_delta_p));
    SAFE_CUDA(hipFree(d_gp_x_grid));
    SAFE_CUDA(hipFree(d_gp_y_grid));
    SAFE_CUDA(hipFree(d_gp_is_not_border));
}

__global__ void gpu_compute_g(double *g, double *r, double alpha, int numElements) {
  int no_thread = threadIdx.x + blockDim.x * blockIdx.x;

  if (no_thread < numElements) {
  	g[no_thread] = r[no_thread] - alpha * g[no_thread];
  }
}

void compute_g(GridParameters gp, double *g, double *r, double alpha) {
	int size = gp.get_num_x_points() * gp.get_num_y_points() * sizeof(double);
    int numElements = gp.get_num_x_points() * gp.get_num_y_points();

	double *d_g, *d_r;
	SAFE_CUDA(hipMalloc(&d_g, size));
	SAFE_CUDA(hipMalloc(&d_r, size));

	SAFE_CUDA(hipMemcpy(d_g, g, size, hipMemcpyHostToDevice));
	SAFE_CUDA(hipMemcpy(d_r, r, size, hipMemcpyHostToDevice));

	int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    gpu_compute_g<<<blocksPerGrid, threadsPerBlock>>>(d_g, d_r, alpha, numElements); CUDA_CHECK_ERROR;

    SAFE_CUDA(hipMemcpy(g, d_g, size, hipMemcpyDeviceToHost));

    SAFE_CUDA(hipFree(d_g));
    SAFE_CUDA(hipFree(d_r));
}

__global__ void gpu_compute_p(double *p, double *p_prev, double *g, double tau, int numElements) {
  int no_thread = threadIdx.x + blockDim.x * blockIdx.x;

  if (no_thread < numElements) {
  	p[no_thread] = p_prev[no_thread] - tau * g[no_thread];
  }
}


void compute_p(GridParameters gp, double *p, double* p_prev, double *g, double tau) {
	int size = gp.get_num_x_points() * gp.get_num_y_points() * sizeof(double);
    int numElements = gp.get_num_x_points() * gp.get_num_y_points();

	double *d_p, *d_p_prev, *d_g;
	SAFE_CUDA(hipMalloc(&d_p, size));
	SAFE_CUDA(hipMalloc(&d_p_prev, size));
	SAFE_CUDA(hipMalloc(&d_g, size));

	SAFE_CUDA(hipMemcpy(d_p_prev, p_prev, size, hipMemcpyHostToDevice));
	SAFE_CUDA(hipMemcpy(d_g, g, size, hipMemcpyHostToDevice));

	int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    gpu_compute_p<<<blocksPerGrid, threadsPerBlock>>>(d_p, d_p_prev, d_g,  tau, numElements); CUDA_CHECK_ERROR;

    SAFE_CUDA(hipMemcpy(p, d_p, size, hipMemcpyDeviceToHost));

    SAFE_CUDA(hipFree(d_p));
    SAFE_CUDA(hipFree(d_p_prev));
    SAFE_CUDA(hipFree(d_g));
}

__global__ void gpu_norm(double *p, double *p_prev, double *results, int n) {
	extern __shared__ double sdata[];
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	// load input into __shared__ memory
	double x = 0.0;
	if(i < n)
		x = abs(p[i]-p_prev[i]);

	sdata[tx] = x;
	__syncthreads(); 
	// block-wide reduction in __shared__ mem
	for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
		if(tx < offset) {
			// add a partial sum upstream to our own
			sdata[tx] = max(sdata[tx], sdata[tx + offset]);
		}
		__syncthreads();
	}
	// finally, thread 0 writes the result
	if(threadIdx.x == 0) {
		// note that the result is per-block
		// not per-thread
		results[blockIdx.x] = sdata[0];
	}
}


__global__ void gpu_reduce_max(double *input, double *results, int n) {
	extern __shared__ double sdata[];
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	// load input into __shared__ memory
	double x = 0.0;
	if(i < n)
		x = input[i];

	sdata[tx] = x;
	__syncthreads(); 
	// block-wide reduction in __shared__ mem
	for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
		if(tx < offset) {
			// add a partial sum upstream to our own
			sdata[tx] = max(sdata[tx], sdata[tx + offset]);
		}
		__syncthreads();
	}
	// finally, thread 0 writes the result
	if(threadIdx.x == 0) {
		// note that the result is per-block
		// not per-thread
		results[blockIdx.x] = sdata[0];
	}
}

double compute_norm(GridParameters gp, double *p, double *p_prev) {
	int size = gp.get_num_x_points() * gp.get_num_y_points() * sizeof(double);
    int numElements = gp.get_num_x_points() * gp.get_num_y_points();

	double *d_p, *d_p_prev, *d_norm;
	SAFE_CUDA(hipMalloc(&d_p, size));
	SAFE_CUDA(hipMalloc(&d_p_prev, size));

	SAFE_CUDA(hipMemcpy(d_p, p, size, hipMemcpyHostToDevice));
	SAFE_CUDA(hipMemcpy(d_p_prev, p_prev, size, hipMemcpyHostToDevice));

	int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	SAFE_CUDA(hipMalloc(&d_norm, (blocksPerGrid+1)*sizeof(double)));
    //printf("rank=%d CUDA kernel launch with %d blocks of %d threads numElements=%d\n", gp.rank, blocksPerGrid, threadsPerBlock, numElements);
    // reduce per-block partial sums 
    gpu_norm<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(double)>>>(d_p, d_p_prev, d_norm, numElements); CUDA_CHECK_ERROR;

    // reduce partial sums to a total sum
    gpu_reduce_max<<<1, threadsPerBlock, threadsPerBlock * sizeof(double)>>>(d_norm, d_norm+blocksPerGrid, blocksPerGrid); CUDA_CHECK_ERROR;

    double gpu_norm = 0.0;
    SAFE_CUDA(hipMemcpy(&gpu_norm, d_norm+blocksPerGrid, sizeof(double), hipMemcpyDeviceToHost));
    //printf("rank=%d size=%d numElements=%d blocksPerGrid=%d CPU product=%f GPU product=%f\n", 
    //	gp.rank, size, numElements, blocksPerGrid, product, gpu_product);

    SAFE_CUDA(hipFree(d_p));
    SAFE_CUDA(hipFree(d_p_prev));
    SAFE_CUDA(hipFree(d_norm));

	double global_norm = 0.0;
	int status = MPI_Allreduce(&gpu_norm, &global_norm, 1, MPI_DOUBLE, MPI_MAX, gp.comm);
    if (status != MPI_SUCCESS) throw std::runtime_error("Error in compute scalar_product!");
    //printf("rank %d: norm=%f global_norm=%f\n", gp.rank, norm, global_norm);
    return global_norm;
}

void init_vector(GridParameters gp, double* func) {
	int i, j;

	for (i=0; i<gp.get_num_x_points(); i++) {
    	for (j=0; j<gp.get_num_y_points(); j++) {
    		int grid_i, grid_j;
    		gp.get_real_grid_index(i, j, grid_i, grid_j);
    		func[i*gp.get_num_y_points()+j] = 0.0;
		}
	}
}

void init_p_prev(GridParameters gp, double* p_prev) {
	int i, j;

	for (i=0; i<gp.get_num_x_points(); i++) {
    	for (j=0; j<gp.get_num_y_points(); j++) {
    		int grid_i, grid_j;
    		gp.get_real_grid_index(i, j, grid_i, grid_j);
    		if (not gp.is_border_point(grid_i, grid_j)) {
                p_prev[i*gp.get_num_y_points()+j] = 0.0;
            }
            else {
                p_prev[i*gp.get_num_y_points()+j] = phi(gp.get_x_grid_value(grid_i), gp.get_y_grid_value(grid_j));
            }
		}
	}
}


int main (int argc, char** argv) {
	if (argc != 3)
		throw std::runtime_error("Incorrect number of arguments");
	clock_t begin = clock();

	const double A1 = 0.0;
	const double A2 = 3.0;
	const double B1 = 0.0;
	const double B2 = 3.0;

	const int N1 = atoi(argv[1]);
	const int N2 = atoi(argv[2]);
	const double eps = 0.0001;

	double* x_grid = new double [N1+1];
	double* y_grid = new double [N2+1];

	for (int i=0; i<=N1; i++) {
		x_grid[i] = A2 * f_grid(1.0*i/N1) + A1 * (1 - f_grid(1.0*i/N1));
		//std::cout << "x_grid[" << i << "]=" << x_grid[i] << std::endl;
	}
	for (int j=0; j<=N2; j++) {
		y_grid[j] = B2 * f_grid(1.0*j/N2) + B1 * (1 - f_grid(1.0*j/N2));
		//std::cout << "y_grid[" << j << "]=" << y_grid[j] << std::endl;
	}

	int rank, size;
	int p1, p2;

	MPI_Init (&argc, &argv);	/* starts MPI */
	MPI_Comm_rank (MPI_COMM_WORLD, &rank);	/* get current process id */
	MPI_Comm_size (MPI_COMM_WORLD, &size);	/* get number of processes */

	compute_grid_processes_number(size, p1, p2);

	// filter extra processes
	if (rank < p1 * p2) {
		if (rank == 0) {
			std::cout << "p1=" << p1 << " p2=" << p2 << " size=" << size << std::endl;
	    }

	    GridParameters gp(rank, MPI_COMM_WORLD, x_grid, y_grid, N1, N2, p1, p2, eps);
	   	//printf("rank %d: x_index_from = %d  x_index_to = %d  y_index_from = %d y_index_to = %d  top=%d bottom=%d left=%d right=%d\n", 
	    //	gp.rank, gp.x_index_from, gp.x_index_to, gp.y_index_from, gp.y_index_to, gp.top, gp.bottom, gp.left, gp.right);
	    
	    double* p = new double [gp.get_num_x_points() * gp.get_num_y_points()];
	    double* p_prev = new double [gp.get_num_x_points() * gp.get_num_y_points()];
	    double* g = new double [gp.get_num_x_points() * gp.get_num_y_points()];
	    double* r = new double [gp.get_num_x_points() * gp.get_num_y_points()];
	    double* delta_p = new double [gp.get_num_x_points() * gp.get_num_y_points()];
	    double* delta_r = new double [gp.get_num_x_points() * gp.get_num_y_points()];
	    double* delta_g = new double [gp.get_num_x_points() * gp.get_num_y_points()];
	    
	    init_p_prev(gp, p_prev);

	    init_vector(gp, r);
	    init_vector(gp, g);
	    init_vector(gp, delta_p);
	    init_vector(gp, delta_g);
	    init_vector(gp, delta_r);

	    double scalar_product_delta_g_and_g = 1.0;
	    double scalar_product_delta_r_and_g = 1.0;
	    double scalar_product_r_and_g = 1.0;
	    double alpha = 0.0;
	    double tau = 0.0;

	    double* phi_on_grid = new double [gp.get_num_x_points() * gp.get_num_y_points()];
	    for (int i=0; i<gp.get_num_x_points(); i++) {
	    	for (int j=0; j<gp.get_num_y_points(); j++) {
	    		int grid_i, grid_j;
	    		gp.get_real_grid_index(i, j, grid_i, grid_j);
	    		phi_on_grid[i*gp.get_num_y_points()+j] = phi(gp.get_x_grid_value(grid_i), gp.get_y_grid_value(grid_j));
	    	}
		}

	    int n_iter = 1;
	    while (true) {
	    	compute_approx_delta(gp, delta_p, p_prev);
	    	compute_r(gp, r, delta_p);

	    	if (n_iter > 1) {
	    		compute_approx_delta(gp, delta_r, r);
	    		scalar_product_delta_r_and_g = scalar_product(gp, delta_r, g);
	    		alpha = 1.0 * scalar_product_delta_r_and_g / scalar_product_delta_g_and_g;
	    	}

	    	if (n_iter > 1) 
	    		compute_g(gp, g, r, alpha);
	    	else 
            	swap(g, r);

            compute_approx_delta(gp, delta_g, g);
            if (n_iter > 1) {
            	scalar_product_r_and_g = scalar_product(gp, r, g);
            }
            else {
            	scalar_product_r_and_g = scalar_product(gp, g, g);
            }

            scalar_product_delta_g_and_g = scalar_product(gp, delta_g, g);
	        tau = 1.0 * scalar_product_r_and_g / scalar_product_delta_g_and_g;

	       	compute_p(gp, p, p_prev, g, tau);
	       	double norm_p_prev = compute_norm(gp, p, p_prev);
	       	double norm_p_phi = compute_norm(gp, p, phi_on_grid);
	       	if (rank == 0)
	       		printf("# iteration %d: norm_p_p_prev=%f norm_p_phi=%f\n", n_iter, norm_p_prev, norm_p_phi);
	       	if (norm_p_prev < gp.eps)
            	break;

            swap(p, p_prev);
	    	n_iter += 1;
	    }
	}
	MPI_Finalize();

	if (rank == 0) {
		clock_t end = clock();
	  	double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
	  	printf("Algorithm finished! Elapsed time: %f sec\n", elapsed_secs);
	}
	return 0;
}
